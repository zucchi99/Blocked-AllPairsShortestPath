#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <cassert>

#include "hip/hip_runtime.h"
#include ""

#include "include/cuda_errors_utils.cuh"
#include "include/adj_matrix_utils.hpp"
#include "include/host_floyd_warshall.hpp"
#include "include/statistical_test.hpp"
#include "include/num_macro.hpp"

#define MAX_BLOCK_SIZE 1024 // in realtà basta fare le proprerties della macchina

__global__ void execute_round_device_v1_1(int *matrix, int n, int t, int row, int col, int B);
void floyd_warshall_blocked_device_v1_1(int *matrix, int n, int B);

int main() {

    multi_size_statistical_test(&floyd_warshall_blocked_device_v1_1, 8, 128, 8, 32, 100, RANDOM_SEED, true, true);

    return 0;
}

__global__ void execute_round_device_v1_1(int *matrix, int n, int t, int row, int col, int B) {

    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

    int i = tid_x + row * B;  // row
    int j = tid_y + col * B;  // col

    //foreach k: t*B <= t < t+B
    for (int k = t * B; k < (t+1) * B; k++) {

        int a, b;
        // i>=row * B && i<(row+1) * B && j>=col * B && j<(col+1) * B;

        // check if thread correspond to one of the cells in current block
        // if (run_this) {

        a = matrix[i*n + j];
        b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 
        // }

        // __syncthreads();

        if (b < a) {
            matrix[i*n + j] = b;
        }
        
        __syncthreads();

    }
}

void floyd_warshall_blocked_device_v1_1(int *matrix, int n, int B) {

    assert(n%B == 0);                       // B must divide n
    assert(B*B<=MAX_BLOCK_SIZE);            // B*B cannot exceed max block size

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));

    int num_rounds = n/B;
     
    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        dim3 num_blocks_phase_1(1, 1);
        dim3 threads_per_block_phase_1(B, B);

        execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, t, t, B);
        HANDLE_ERROR(hipDeviceSynchronize());

        //phase 2 blocks left
        for (int j = t-1; j >= 0; j--) {
            execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, t, j, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        //phase 2 blocks above
        for (int i = t-1; i >= 0; i--) {
            execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, i, t, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        //phase 2 blocks below
        for (int i = t+1; i < num_rounds; i++) {
            execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, i, t, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        //phase 2 blocks right
        for (int j = t+1; j < num_rounds; j++) {
            execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, t, j, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        HANDLE_ERROR(hipDeviceSynchronize());
        
        //phase 3 blocks above and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }
        //phase 3 blocks above and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }
        //phase 3 blocks below and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }      
        //phase 3 blocks below and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device_v1_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }

        HANDLE_ERROR(hipDeviceSynchronize());   
    }

    // HANDLE_ERROR(hipDeviceSynchronize());  

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_rand_matrix));
}

