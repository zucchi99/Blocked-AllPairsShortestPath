#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cassert>

#include "hip/hip_runtime.h"
#include ""

#include "include/adj_matrix_utils.cuh"
#include "include/adj_matrix_utils.hpp"
#include "include/cuda_errors_utils.cuh"
#include "include/host_floyd_warshall.hpp"
#include "include/macros.hpp"
#include "include/performance_test.cuh"
#include "include/statistical_test.hpp"

//main device code
void floyd_warshall_blocked_device_v_2_0(int *matrix, int n, int B);

//rounds code
__global__ void execute_round_device_v_2_0_phase_1(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_2_0_phase_2_row(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_2_0_phase_2_col(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_2_0_phase_3(int *matrix, int n, int t, int B);

__global__ void execute_round_device_v_1_4_phase_2_row(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  L1  L2  L3  R1  R2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    int i, j;

    // it's a row ...
    i = BLOCK_START(t, B) + threadIdx.x;

    if (blockIdx.y < t) {

        // ... and it's the left one
        j = BLOCK_START(blockIdx.y, B) + threadIdx.y;

    } else {
        
        // ... and it's the right one
        j = BLOCK_START(blockIdx.y, B) + B + threadIdx.y;
    }

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
            matrix[i*n + j] = b;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();

    }
}

__global__ void execute_round_device_v_1_4_phase_2_col(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  U1  U2  U3  D1  D2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    int i, j;

    // it's a column ...
    j = BLOCK_START(t, B) + threadIdx.y;

    if (blockIdx.y < t) {

        // ... and it's the up one
        i = BLOCK_START(blockIdx.y, B) + threadIdx.x;

    } else {

        // ... and it's the down one
        i = BLOCK_START(blockIdx.y, B) + B + threadIdx.x;
    }

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
            matrix[i*n + j] = b;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();
    }
}


int main() {

    multi_size_statistical_test(&floyd_warshall_blocked_device_v_2_0, 8, 256, 8, 32, 100, RANDOM_SEED, false, true);
    
    // int n = 256;
    // int B = 32;
    //int *test_instance = allocate_arr_matrix(n, n);
    //int *input_instance = allocate_arr_matrix(n, n);
    // int rand_seed = time(NULL);
    // printf("rand_seed: %d\n", rand_seed);
    //populate_arr_adj_matrix(input_instance, n, rand_seed, false);
    //do_nvprof_performance_test(&floyd_warshall_blocked_device_v_2_0, n, B, 10, rand_seed);

    //single test
    /*
    size_t n = 6;
    int BLOCKING_FACTOR = 2;
    printf("n: %ld, B: %d\n", n, BLOCKING_FACTOR);
    int n_err = do_arr_floyd_warshall_statistical_test(&floyd_warshall_blocked_device_v_2_0, n, BLOCKING_FACTOR, 1, RANDOM_SEED, true, 4, true);
    printf("n_err:%d\n", n_err);
    */

    return 0;
}

void floyd_warshall_blocked_device_v_2_0(int *matrix, int n, int B) {

    assert(n%B == 0);                       // B must divide n
    assert(B*B<=MAX_BLOCK_SIZE);            // B*B cannot exceed max block size

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));

    int num_rounds = n/B;
     
    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        dim3 num_blocks_phase_1(1, 1);
        dim3 threads_per_block_phase_1(B, B);

        execute_round_device_v_2_0_phase_1<<<num_blocks_phase_1, threads_per_block_phase_1, B*B*sizeof(int)>>>(dev_rand_matrix, n, t, B);
        HANDLE_ERROR(hipDeviceSynchronize());

        // phase 2: all blocks that share a row or a column with the self dependent, so
        //  -   all blocks just above or under t
        //  -   all block at left and at right of t

        dim3 num_blocks_phase_2(1, num_rounds-1);  

        execute_round_device_v_2_0_phase_2_row<<<num_blocks_phase_2, threads_per_block_phase_1, 2*B*B*sizeof(int)>>>(dev_rand_matrix, n, t, B);
        execute_round_device_v_2_0_phase_2_col<<<num_blocks_phase_2, threads_per_block_phase_1, 2*B*B*sizeof(int)>>>(dev_rand_matrix, n, t, B);

        // execute_round_device_v_1_4_phase_2_row<<<num_blocks_phase_2, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        // execute_round_device_v_1_4_phase_2_col<<<num_blocks_phase_2, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);


        HANDLE_ERROR(hipDeviceSynchronize());

        // phase 3: all the remaining blocks, so all the blocks that don't share a row or a col with t

        dim3 num_blocks_phase_3(num_rounds-1, num_rounds-1); 

        execute_round_device_v_2_0_phase_3<<<num_blocks_phase_3, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        HANDLE_ERROR(hipDeviceSynchronize()); 
    }

    // HANDLE_ERROR(hipDeviceSynchronize());  

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(dev_rand_matrix));
}


__global__ void execute_round_device_v_2_0_phase_1(int *matrix, int n, int t, int B) {

    // Launched block and correspondent position in the matrix

    //  t

    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   t   .   .
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 

    extern __shared__ int block_t_t_shared[];

    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

    int i = tid_x + t * B;  // row
    int j = tid_y + t * B;  // col

    block_t_t_shared[tid_x*B + tid_y] = matrix[i*n + j];

    __syncthreads();

    //foreach k: t*B <= t < t+B
    for (int k = 0; k < B; k++) {

        int b = sum_if_not_infinite(block_t_t_shared[tid_x*B + k], block_t_t_shared[k*B + tid_y], INF); 

        if (b < block_t_t_shared[tid_x*B + tid_y]) {
            block_t_t_shared[tid_x*B + tid_y] = b;
        }
        
        __syncthreads();
    }

    matrix[i*n + j] = block_t_t_shared[tid_x*B + tid_y];
}

__global__ void execute_round_device_v_2_0_phase_2_row(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  L1  L2  L3  R1  R2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    extern __shared__ int shared_mem[];
    
    int* block_t_t_shared = &shared_mem[0];
    int* block_i_j_shared = &shared_mem[B*B];

    int i, j;

    // it's a row ...
    i = BLOCK_START(t, B) + threadIdx.x;

    if (blockIdx.y < t) {

        // ... and it's the left one
        j = BLOCK_START(blockIdx.y, B) + threadIdx.y;

    } else {
        
        // ... and it's the right one
        j = BLOCK_START(blockIdx.y, B) + B + threadIdx.y;
    }

    block_i_j_shared[threadIdx.x*B + threadIdx.y] = matrix[i*n + j];

    block_t_t_shared[threadIdx.x*B + threadIdx.y] = matrix[
        (BLOCK_START(t, B) + threadIdx.x) * n
        + (BLOCK_START(t, B) + threadIdx.y)
    ];

    __syncthreads();

    //foreach k: t*B <= t < t+B
    for (int k = 0; k < B; k++) {

        // Because we are doing rows:
        // -    matrix[i,k] is in block_t_t_shared[threadIdx.x,k]
        // -    matrix[k,j] is in block_i_j_shared[k,threadIdx.y]
        int b = sum_if_not_infinite(block_t_t_shared[threadIdx.x*B + k], block_i_j_shared[k*B + threadIdx.y], INF); 

        if (b < block_i_j_shared[threadIdx.x*B + threadIdx.y]) {
            block_i_j_shared[threadIdx.x*B + threadIdx.y] = b;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();
    }

    // copy result in global memory
    matrix[i*n + j] = block_i_j_shared[threadIdx.x*B + threadIdx.y];
}

__global__ void execute_round_device_v_2_0_phase_2_col(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  U1  U2  U3  D1  D2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    extern __shared__ int shared_mem[];

    int* block_i_j_shared = &shared_mem[0];
    int* block_t_t_shared = &shared_mem[B*B];

    int i, j;

    // it's a column ...
    j = BLOCK_START(t, B) + threadIdx.y;

    if (blockIdx.y < t) {

        // ... and it's the up one
        i = BLOCK_START(blockIdx.y, B) + threadIdx.x;

    } else {

        // ... and it's the down one
        i = BLOCK_START(blockIdx.y, B) + B + threadIdx.x;
    }

    block_i_j_shared[threadIdx.x*B + threadIdx.y] = matrix[i*n + j];

    block_t_t_shared[threadIdx.x*B + threadIdx.y] = matrix[
        (BLOCK_START(t, B) + threadIdx.x) * n
        + (BLOCK_START(t, B) + threadIdx.y)
    ];

    //foreach k: t*B <= t < t+B
    for (int k = 0; k < B; k++) {
        
        // Because we are doing columns:
        // -    matrix[i,k] is in block_i_j_shared[threadIdx.x,k]
        // -    matrix[k,j] is in block_t_t_shared[k,threadIdx.y]
        int b = sum_if_not_infinite(block_i_j_shared[threadIdx.x*B + k], block_t_t_shared[k*B + threadIdx.y], INF); 

        if (b < block_i_j_shared[threadIdx.x*B + threadIdx.y]) {
            block_i_j_shared[threadIdx.x*B + threadIdx.y] = b;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();
    }

    // copy result in global memory
    matrix[i*n + j] = block_i_j_shared[threadIdx.x*B + threadIdx.y];
}


__global__ void execute_round_device_v_2_0_phase_3(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix

    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  DL  DL  DL  DR  DR
    //  DL  DL  DL  DR  DR

    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR  
    //  -   -   -   -   -   - 
    //  DL  DL  DL  -   DR  DR
    //  DL  DL  DL  -   DR  DR

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
           
    // if a thread is under t, add B as row offset to get right position in matrix
    if (blockIdx.x >= t)    i += B; 

    // if a thread is ar right of t, add B as col offset to get right position in matrix
    if (blockIdx.y >= t)    j += B;

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
                matrix[i*n + j] = b;
        }

        __syncthreads();
    }
}

