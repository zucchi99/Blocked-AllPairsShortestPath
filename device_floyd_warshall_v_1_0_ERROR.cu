#include "hip/hip_runtime.h"

#include "include/include_needed_libraries.cuh"

//main device code
void floyd_warshall_blocked_device_v_1_0(int *matrix, int n, int B);

//rounds code
__global__ void execute_round_device_v_1_0(int *matrix, int n, int t, int row, int col, int B);

int main(int argc, char *argv[]) {

    return handle_arguments_and_execute(argc, argv, (void(*) (int*, int, int)) &floyd_warshall_blocked_device_v_1_0);

}

__global__ void execute_round_device_v_1_0(int *matrix, int n, int t, int row, int col, int B) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int i = tid/n;  // row
    int j = tid%n;  // col

    //foreach k: t*B <= t < t+B
    for (int k = t * B; k < (t+1) * B; k++) {

        int a, b;
        bool run_this = ((i >= row*B) && (i < (row+1)*B) && (j >= col*B) && (j < (col+1)*B));

        // check if thread correspond to one of the cells in current block
        if (run_this) {

            // WARNING: do NOT put the macro directly into 
            a = matrix[i*n + j];
            b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 
        }

        __syncthreads();


        if (run_this) {
            matrix[i*n + j] = mmin(a, b);
        }
        
        __syncthreads();

    }
}


void floyd_warshall_blocked_device_v_1_0(int *matrix, int n, int B) {

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));
    
    int num_rounds = n/B;

    int num_blocks = num_rounds*num_rounds;
    int thread_per_block = B*B; 
    

    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, t, B);
        HANDLE_ERROR(hipDeviceSynchronize());

        //phase 2 blocks left
        for (int j = t-1; j >= 0; j--) {
            execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        //phase 2 blocks above
        for (int i = t-1; i >= 0; i--) {
            execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        //phase 2 blocks below
        for (int i = t+1; i < num_rounds; i++) {
            execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        //phase 2 blocks right
        for (int j = t+1; j < num_rounds; j++) {
            execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
            // HANDLE_ERROR(hipDeviceSynchronize());  
        }

        HANDLE_ERROR(hipDeviceSynchronize());
        
        //phase 3 blocks above and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }
        //phase 3 blocks above and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }
        //phase 3 blocks below and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }      
        //phase 3 blocks below and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device_v_1_0<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
                // HANDLE_ERROR(hipDeviceSynchronize());  
            }
        }

        // HANDLE_ERROR(hipDeviceSynchronize());   
    }

    HANDLE_ERROR(hipDeviceSynchronize());  

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_rand_matrix));
}

