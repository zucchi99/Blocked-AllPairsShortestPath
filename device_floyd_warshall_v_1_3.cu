#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cassert>

#include "hip/hip_runtime.h"
#include ""

#include "include/adj_matrix_utils.cuh"
#include "include/adj_matrix_utils.hpp"
#include "include/cuda_errors_utils.cuh"
#include "include/host_floyd_warshall.hpp"
#include "include/macros.hpp"
#include "include/statistical_test.hpp"

//main device code
void floyd_warshall_blocked_device_v_1_3(int *matrix, int n, int B);

//rounds code
__global__ void execute_round_device_v_1_2_phase_1(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_1_3_phase_2(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_1_3_phase_3(int *matrix, int n, int t, int B);

int main() {

    multi_size_statistical_test(&floyd_warshall_blocked_device_v_1_3, 8, 256, 8, 32, 1000, RANDOM_SEED, false, true);

    //single test
    /*
    size_t n = 6;
    int BLOCKING_FACTOR = 2;
    printf("n: %ld, B: %d\n", n, BLOCKING_FACTOR);
    int n_err = do_arr_floyd_warshall_statistical_test(&floyd_warshall_blocked_device_v_1_3, n, BLOCKING_FACTOR, 1, RANDOM_SEED, true, 4, true);
    printf("n_err:%d\n", n_err);
    */

    return 0;
}

void floyd_warshall_blocked_device_v_1_3(int *matrix, int n, int B) {

    assert(n%B == 0);                       // B must divide n
    assert(B*B<=MAX_BLOCK_SIZE);            // B*B cannot exceed max block size

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));

    int num_rounds = n/B;
     
    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        dim3 num_blocks_phase_1(1, 1);
        dim3 threads_per_block_phase_1(B, B);

        execute_round_device_v_1_2_phase_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        HANDLE_ERROR(hipDeviceSynchronize());

        // phase 2: all blocks that share a row or a column with the self dependent, so
        //  -   all blocks just above or under t
        //  -   all block at left and at right of t

        dim3 num_blocks_phase_2(2, num_rounds-1);  

        execute_round_device_v_1_3_phase_2<<<num_blocks_phase_2, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        HANDLE_ERROR(hipDeviceSynchronize());

        // phase 3: all the remaining blocks, so all the blocks that don't share a row or a col with t

        dim3 num_blocks_phase_3(num_rounds-1, num_rounds-1); 

        execute_round_device_v_1_3_phase_3<<<num_blocks_phase_3, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        HANDLE_ERROR(hipDeviceSynchronize()); 
    }

    // HANDLE_ERROR(hipDeviceSynchronize());  

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(dev_rand_matrix));
}


__global__ void execute_round_device_v_1_2_phase_1(int *matrix, int n, int t, int B) {

    // Launched block and correspondent position in the matrix

    //  t

    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   t   .   .
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 

    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

    int i = tid_x + t * B;  // row
    int j = tid_y + t * B;  // col

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
            matrix[i*n + j] = b;
        }
        
        __syncthreads();
    }
}

__global__ void execute_round_device_v_1_3_phase_2(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  L1  L2  L3  R1  R2
    //  U1  U2  U3  D1  D2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    int i, j;

    if (blockIdx.x == 0) {  

        // it's a row ...
        i = BLOCK_START(t, B) + threadIdx.x;

        if (blockIdx.y < t) {

            // ... and it's the left one
            j = BLOCK_START(blockIdx.y, B) + threadIdx.y;

        } else {
            
            // ... and it's the right one
            j = BLOCK_START(blockIdx.y, B) + B + threadIdx.y;
        }
    } else {

        // it's a column ...
        j = BLOCK_START(t, B) + threadIdx.y;

        if (blockIdx.y < t) {

            // ... and it's the up one
            i = BLOCK_START(blockIdx.y, B) + threadIdx.x;

        } else {

            // ... and it's the down one
            i = BLOCK_START(blockIdx.y, B) + B + threadIdx.x;
        }
    }

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        if (
            /* row index is contained in s.d. block and column index is outside */
            ( BLOCK_START(t,B)<=i<BLOCK_END(t,B) && (j<BLOCK_START(t,B) || j>=BLOCK_END(t,B)) )   ||  

            /* column index is contained in s.d. block and row index is outside */
            ( BLOCK_START(t,B)<=j<BLOCK_END(t,B) && (i<BLOCK_START(t,B) || i>=BLOCK_END(t,B)) ) 
            ) {

            int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

            if (b < matrix[i*n + j]) {
                matrix[i*n + j] = b;
            }
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();

    }
}


__global__ void execute_round_device_v_1_3_phase_3(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix

    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  DL  DL  DL  DR  DR
    //  DL  DL  DL  DR  DR

    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR  
    //  -   -   -   -   -   - 
    //  DL  DL  DL  -   DR  DR
    //  DL  DL  DL  -   DR  DR

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
           
    // if a thread is under t, add B as row offset to get right position in matrix
    if (blockIdx.x >= t)    i += B; 

    // if a thread is ar right of t, add B as col offset to get right position in matrix
    if (blockIdx.y >= t)    j += B;

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
                matrix[i*n + j] = b;
        }

        __syncthreads();
    }
}

