#include "hip/hip_runtime.h"
#include "include/include_needed_libraries.cuh"

//main device code
void floyd_warshall_blocked_device_v_1_4(int *matrix, int n, int B);

//rounds code
__global__ void execute_round_device_v_1_4_phase_1(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_1_4_phase_2_row(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_1_4_phase_2_col(int *matrix, int n, int t, int B);
__global__ void execute_round_device_v_1_4_phase_3(int *matrix, int n, int t, int B);

int main(int argc, char *argv[]) {

    return handle_arguments_and_execute(argc, argv, (void(*) (int*, int, int)) &floyd_warshall_blocked_device_v_1_4);

}

void floyd_warshall_blocked_device_v_1_4(int *matrix, int n, int B) {

    assert(n%B == 0);                       // B must divide n
    assert(B*B<=MAX_BLOCK_SIZE);            // B*B cannot exceed mmax block size

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));

    int num_rounds = n/B;
     
    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        dim3 num_blocks_phase_1(1, 1);
        dim3 threads_per_block_phase_1(B, B);

        execute_round_device_v_1_4_phase_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        HANDLE_ERROR(hipDeviceSynchronize());

        // phase 2: all blocks that share a row or a column with the self dependent, so
        //  -   all blocks just above or under t
        //  -   all block at left and at right of t

        dim3 num_blocks_phase_2(1, num_rounds-1);  

        execute_round_device_v_1_4_phase_2_row<<<num_blocks_phase_2, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        execute_round_device_v_1_4_phase_2_col<<<num_blocks_phase_2, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);

        HANDLE_ERROR(hipDeviceSynchronize());

        // phase 3: all the remaining blocks, so all the blocks that don't share a row or a col with t

        dim3 num_blocks_phase_3(num_rounds-1, num_rounds-1); 

        execute_round_device_v_1_4_phase_3<<<num_blocks_phase_3, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B);
        HANDLE_ERROR(hipDeviceSynchronize()); 
    }

    // HANDLE_ERROR(hipDeviceSynchronize());  

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(dev_rand_matrix));
}


__global__ void execute_round_device_v_1_4_phase_1(int *matrix, int n, int t, int B) {

    // Launched block and correspondent position in the matrix

    //  t

    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   t   .   .
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 

    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

    int i = tid_x + t * B;  // row
    int j = tid_y + t * B;  // col

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
            matrix[i*n + j] = b;
        }
        
        __syncthreads();
    }
}

__global__ void execute_round_device_v_1_4_phase_2_row(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  L1  L2  L3  R1  R2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    int i, j;

    // it's a row ...
    i = BLOCK_START(t, B) + threadIdx.x;

    if (blockIdx.y < t) {

        // ... and it's the left one
        j = BLOCK_START(blockIdx.y, B) + threadIdx.y;

    } else {
        
        // ... and it's the right one
        j = BLOCK_START(blockIdx.y, B) + B + threadIdx.y;
    }

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
            matrix[i*n + j] = b;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();

    }
}

__global__ void execute_round_device_v_1_4_phase_2_col(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  U1  U2  U3  D1  D2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    int i, j;

    // it's a column ...
    j = BLOCK_START(t, B) + threadIdx.y;

    if (blockIdx.y < t) {

        // ... and it's the up one
        i = BLOCK_START(blockIdx.y, B) + threadIdx.x;

    } else {

        // ... and it's the down one
        i = BLOCK_START(blockIdx.y, B) + B + threadIdx.x;
    }

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (b < matrix[i*n + j]) {
            matrix[i*n + j] = b;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();
    }
}


__global__ void execute_round_device_v_1_4_phase_3(int *matrix, int n, int t, int B) {

    // Launched blocks and correspondent position in the matrix

    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  DL  DL  DL  DR  DR
    //  DL  DL  DL  DR  DR

    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR  
    //  -   -   -   -   -   - 
    //  DL  DL  DL  -   DR  DR
    //  DL  DL  DL  -   DR  DR

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
           
    // if a thread is under t, add B as row offset to get right position in matrix
    if (blockIdx.x >= t)    i += B; 

    // if a thread is ar right of t, add B as col offset to get right position in matrix
    if (blockIdx.y >= t)    j += B;

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int using_k_path = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 

        if (using_k_path < matrix[i*n + j]) {
            matrix[i*n + j] = using_k_path;
        }

        __syncthreads();
    }
}

