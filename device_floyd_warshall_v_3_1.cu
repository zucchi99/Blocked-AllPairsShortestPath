#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cassert>
#include <vector>

#include "hip/hip_runtime.h"
#include ""

#include "include/adj_matrix_utils.cuh"
#include "include/adj_matrix_utils.hpp"
#include "include/cuda_errors_utils.cuh"
#include "include/host_floyd_warshall.hpp"
#include "include/macros.hpp"
#include "include/performance_test.cuh"
#include "include/statistical_test.hpp"
#include "include/lcm.hpp"

#define ARR_MATRIX_INDEX(i,j,n) (i*n+j)
#define ARR_MATRIX_INDEX_TRASP(i,j,n) (i+n*j)

#define SHARED_BANK_N_INT 32
#define ARR_MATRIX_INDEX_BANK_CONFLICT(i, j, n, handle_bank_conflict) (i*n + j + (handle_bank_conflict ? i : 0))
#define ARR_MATRIX_SIZE_BANK_CONFICT(B,handle_bank_conflict) (B*B + (handle_bank_conflict ? (B-1) : 0))

//main device code
void floyd_warshall_blocked_device_v_3_1(int *matrix, int n, int B);

//rounds code
__global__ void execute_round_device_v_3_1_phase_1(int *matrix, int n, int t, bool handle_bank_conflict);

__global__ void execute_round_device_v_3_1_phase_2_row_portion(int *matrix, int n, int t, int start_col, int end_col);
__global__ void execute_round_device_v_3_1_phase_2_col_portion(int *matrix, int n, int t, int start_row, int end_row);
__global__ void execute_round_device_v_3_1_phase_3_portion(int *matrix, int n, int t, int start_row, int start_col, int end_row, int end_col);


int main() {

    MultiSizeTestParameters my_params;
    my_params.f = &floyd_warshall_blocked_device_v_3_1;
    my_params.g = &host_array_floyd_warshall_blocked;
    my_params.start_input_size = 4;
    my_params.end_input_size = 100;
    my_params.costant_multiplier = 1.4;
    my_params.min_blocking_factor = 2;

    print_multi_size_test_parameters(my_params);
    multi_size_statistical_test(my_params);

    return 0;
}

hipKernelNodeParams cuda_graph_node_params_copy(hipKernelNodeParams params) {
    
    hipKernelNodeParams newParams = { 0 };

    newParams.func = params.func;
    newParams.blockDim = params.blockDim;
    newParams.gridDim = params.gridDim;
    newParams.kernelParams = params.kernelParams;
    newParams.sharedMemBytes = params.sharedMemBytes;
    newParams.extra = params.extra;

    return newParams;
}


void floyd_warshall_blocked_device_v_3_1(int *matrix, int n, int B) {

    assert(n%B == 0);                       // B must divide n
    assert(B*B<=MAX_BLOCK_SIZE);            // B*B cannot exceed max block size

    hipStream_t streams[4];
    for (int i=0; i<4; i++) {
        hipStreamCreate(&streams[i]);
    }

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n*n*sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));

    int num_rounds = n/B;

    bool bank_conflict_phase_1 = lcm(SHARED_BANK_N_INT, B) <= (B-1)*B;

         
    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        hipGraph_t roundGraph;
        hipGraphCreate(&roundGraph, 0);

        std::vector<hipGraphNode_t> nodeDependencies = {}; // Dependency vector 

        // ----------------------------------------------------------------------
        // phase 1: self-dependent block

        dim3 num_blocks(max(num_rounds-1, 1), max(num_rounds-1, 1));
        dim3 threads_per_block(B, B);

        // execute_round_device_v_3_1_phase_1<<<
        //     num_blocks, 
        //     threads_per_block, 
        //     ARR_MATRIX_SIZE_BANK_CONFICT(B, bank_conflict_phase_1)*sizeof(int), 
        //     streams[0]>>>(dev_rand_matrix, n, t, bank_conflict_phase_1);

        // HANDLE_ERROR(hipDeviceSynchronize());

        void* phase1_args[4] = { (void*) &dev_rand_matrix, (void*) &n, (void*) &t, (void*) &bank_conflict_phase_1 };

        hipKernelNodeParams phase1_params;

        phase1_params.func = (void*) execute_round_device_v_3_1_phase_1;
        phase1_params.gridDim = num_blocks;
        phase1_params.blockDim = threads_per_block;
        phase1_params.sharedMemBytes = max(
            ARR_MATRIX_SIZE_BANK_CONFICT(B, bank_conflict_phase_1)*sizeof(int), 
            2*B*B*sizeof(int)
        );
        phase1_params.kernelParams = (void**) phase1_args;
        phase1_params.extra = NULL;

        hipGraphNode_t phase1_node;

        HANDLE_ERROR(hipGraphAddKernelNode(
            &phase1_node, roundGraph, 
            nodeDependencies.data(), nodeDependencies.size(), 
            &phase1_params));
        

        // ----------------------------------------------------------------------
        // phase 2: row and cols
        // all blocks that share a row or a column with the self dependent, so
        //  -   all blocks just above or under t
        //  -   all block at left and at right of t


        // execute_round_device_v_3_1_phase_2_col_portion<<<
        //     num_blocks, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[0]>>>(dev_rand_matrix, n, t, 0, t);

        // execute_round_device_v_3_1_phase_2_row_portion<<<
        //     num_blocks, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[1]>>>(dev_rand_matrix, n, t, 0, t);

        // execute_round_device_v_3_1_phase_2_col_portion<<<
        //     num_blocks, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[2]>>>(dev_rand_matrix, n, t, t+1, num_rounds);

        // execute_round_device_v_3_1_phase_2_row_portion<<<
        //     num_blocks, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[3]>>>(dev_rand_matrix, n, t, t+1, num_rounds);

        // HANDLE_ERROR(hipDeviceSynchronize()); 

        // nodeDependencies.clear();
        nodeDependencies.push_back(phase1_node);

        // up 
        // execute_round_device_v_3_1_phase_2_col_portion<<<
        //     t, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[0]>>>(dev_rand_matrix, n, t, 0);

        int start_up_left = 0;
        int end_up_left = t;
        void* phase2_up_left_args[5] = { (void*) &dev_rand_matrix, 
            &n, &t, &start_up_left, &end_up_left };

        hipKernelNodeParams phase2_up_params = cuda_graph_node_params_copy(phase1_params);

        phase2_up_params.func = (void*) execute_round_device_v_3_1_phase_2_col_portion;
        // phase2_up_params.sharedMemBytes = 2*B*B*sizeof(int);
        phase2_up_params.kernelParams = (void**) phase2_up_left_args;

        hipGraphNode_t phase2_up_node;

        HANDLE_ERROR(hipGraphAddKernelNode(
            &phase2_up_node, roundGraph, 
            nodeDependencies.data(), nodeDependencies.size(), 
            &phase2_up_params
        ));

        // left
        // execute_round_device_v_3_1_phase_2_row_portion<<<
        //     t, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[1]>>>(dev_rand_matrix, n, t, 0);

        hipKernelNodeParams phase2_left_params = cuda_graph_node_params_copy(phase2_up_params);
        phase2_left_params.func = (void*) execute_round_device_v_3_1_phase_2_row_portion;

        hipGraphNode_t phase2_left_node;

        HANDLE_ERROR(hipGraphAddKernelNode(
            &phase2_left_node, roundGraph, 
            nodeDependencies.data(), nodeDependencies.size(), 
            &phase2_left_params));

        // down
        // execute_round_device_v_3_1_phase_2_col_portion<<<
        //     num_rounds-1-t, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[2]>>>(dev_rand_matrix, n, t, t+1);

        int start_down_right = t+1;
        int end_down_right = num_rounds;

        hipKernelNodeParams phase2_down_params = cuda_graph_node_params_copy(phase2_up_params);
        void* phase2_down_right_args[5] = { (void*) &dev_rand_matrix, 
            &n, &t, &start_down_right, &end_down_right};
        phase2_down_params.kernelParams = (void**) phase2_down_right_args;

        hipGraphNode_t phase2_down_node;

        HANDLE_ERROR(hipGraphAddKernelNode(
            &phase2_down_node, roundGraph, 
            nodeDependencies.data(), nodeDependencies.size(), 
            &phase2_down_params));

        // right
        // execute_round_device_v_3_1_phase_2_row_portion<<<
        //     num_rounds-1-t, threads_per_block, 
        //     2*B*B*sizeof(int), 
        //     streams[3]>>>(dev_rand_matrix, n, t, t+1);

        hipKernelNodeParams phase2_right_params = cuda_graph_node_params_copy(phase2_down_params);
        phase2_right_params.func = (void*) execute_round_device_v_3_1_phase_2_row_portion;

        hipGraphNode_t phase2_right_node;

        HANDLE_ERROR(hipGraphAddKernelNode(
            &phase2_right_node, roundGraph, 
            nodeDependencies.data(), nodeDependencies.size(), 
            &phase2_right_params));

        // HANDLE_ERROR(hipDeviceSynchronize());

        hipGraphExec_t instance;
        
        HANDLE_ERROR(hipGraphInstantiate(&instance, roundGraph, NULL, NULL, 0));

        HANDLE_ERROR(hipGraphLaunch(instance, streams[1]));
        HANDLE_ERROR(hipStreamSynchronize(streams[1]));

        // Clean up
        HANDLE_ERROR(hipGraphExecDestroy(instance));
        HANDLE_ERROR(hipGraphDestroy(roundGraph));

        HANDLE_ERROR(hipDeviceSynchronize()); 

        // phase 3: all the remaining blocks, so all the blocks that don't share a row or a col with t

        // dim3 num_blocks_phase_3(num_rounds-1, num_rounds-1); 
        // execute_round_device_v_3_1_phase_3<<<num_blocks_phase_3, threads_per_block, 2*B*B*sizeof(int)>>>(dev_rand_matrix, n, t);

        dim3 num_blocks_phase_3_ul(t, t);
        execute_round_device_v_3_1_phase_3_portion<<<
            num_blocks, threads_per_block, 
            2*B*B*sizeof(int), 
            streams[0]>>>(dev_rand_matrix, n, t, 0, 0, t, t);

        dim3 num_blocks_phase_3_dr(num_rounds-t-1, num_rounds-t-1); 
        execute_round_device_v_3_1_phase_3_portion<<<
            num_blocks, threads_per_block, 
            2*B*B*sizeof(int), 
            streams[1]>>>(dev_rand_matrix, n, t, t+1, t+1, num_rounds, num_rounds);

        dim3 num_blocks_phase_3_ur(t, num_rounds-t-1); 
        execute_round_device_v_3_1_phase_3_portion<<<
            num_blocks, threads_per_block, 
            2*B*B*sizeof(int), 
            streams[2]>>>(dev_rand_matrix, n, t, 0, t+1, t, num_rounds);

        dim3 num_blocks_phase_3_dl(num_rounds-t-1, t); 
        execute_round_device_v_3_1_phase_3_portion<<<
            num_blocks, threads_per_block, 
            2*B*B*sizeof(int), 
            streams[3]>>>(dev_rand_matrix, n, t, t+1, 0, num_rounds, t);

        HANDLE_ERROR(hipDeviceSynchronize()); 
    }

    // HANDLE_ERROR(hipDeviceSynchronize());  

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(dev_rand_matrix));

    for (int i=0; i<4; i++) {
        HANDLE_ERROR(hipStreamDestroy(streams[i]));
    }
}


__global__ void execute_round_device_v_3_1_phase_1(int *matrix, int n, int t, bool handle_bank_conflict) {

    // Launched block and correspondent position in the matrix

    //  t   -   -   -   -
    //  -   -   -   -   -
    //  -   -   -   -   -
    //  -   -   -   -   -
    //  -   -   -   -   -
    

    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   t   .   .
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 

    if (blockIdx.x > 0 || blockIdx.y > 0)   return;

    // if (threadIdx.x == 0 && threadIdx.y == 0) printf("(%d,%d) ", blockIdx.x, blockIdx.y);

    extern __shared__ int block_t_t_shared[];

    int i = threadIdx.x + t * blockDim.x;  // row abs index
    int j = threadIdx.y + t * blockDim.x;  // col abs index

    block_t_t_shared[ARR_MATRIX_INDEX_BANK_CONFLICT(threadIdx.x, threadIdx.y, blockDim.x, handle_bank_conflict)] = matrix[ARR_MATRIX_INDEX(i, j, n)];

    __syncthreads();

    // now k is iterating the relative indexind of (t,t) block 
    // in shared memory (instead of the abs position in matrix)
    for (int k = 0; k < blockDim.x; k++) {

        int using_k_path = sum_if_not_infinite(
            block_t_t_shared[ARR_MATRIX_INDEX_BANK_CONFLICT(threadIdx.x, k, blockDim.x, handle_bank_conflict)], 
            block_t_t_shared[ARR_MATRIX_INDEX_BANK_CONFLICT(k, threadIdx.y, blockDim.x, handle_bank_conflict)], 
            INF
        ); 

        if (using_k_path < block_t_t_shared[ARR_MATRIX_INDEX_BANK_CONFLICT(threadIdx.x, threadIdx.y, blockDim.x, handle_bank_conflict)]) {
            block_t_t_shared[ARR_MATRIX_INDEX_BANK_CONFLICT(threadIdx.x, threadIdx.y, blockDim.x, handle_bank_conflict)] = using_k_path;
        }
        
        __syncthreads();
    }

    matrix[ARR_MATRIX_INDEX(i, j, n)] = block_t_t_shared[ARR_MATRIX_INDEX_BANK_CONFLICT(threadIdx.x, threadIdx.y, blockDim.x, handle_bank_conflict)];
}


__global__ void execute_round_device_v_3_1_phase_2_row_portion(int *matrix, int n, int t, int start_col, int end_col) {
    
    if (blockIdx.x >= end_col-start_col)    return;
    
    extern __shared__ int shared_mem[];
    
    int* block_i_j_shared = &shared_mem[0];
    int* block_t_t_shared = &shared_mem[(blockDim.x * blockDim.x)];

    // it's a row ...

    // abs row index 
    int i = BLOCK_START(t, blockDim.x) + threadIdx.x;    
    // abs col index   
    int j = BLOCK_START(blockIdx.x, blockDim.x) + threadIdx.y + start_col * blockDim.x; 

    // the block where I am working
    block_i_j_shared[ARR_MATRIX_INDEX(threadIdx.x, threadIdx.y, blockDim.x)] = matrix[ARR_MATRIX_INDEX(i, j, n)];

    // the self-dependent block already calculated in this round (transposed to avoid bank conflict)
    block_t_t_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, threadIdx.y, blockDim.x)] = matrix[
        ARR_MATRIX_INDEX(
            (BLOCK_START(t, blockDim.x) + threadIdx.x), 
            (BLOCK_START(t, blockDim.x) + threadIdx.y), 
            n
        )
    ];


    __syncthreads();

    // now k is iterating the relative indexind of (t,t) block 
    // in shared memory (instead of the abs position in matrix)
    for (int k = 0; k < blockDim.x; k++) {

        // Because we are doing rows:
        // -    matrix[i,abs_k] is in block_t_t_shared[threadIdx.x,k]
        // -    matrix[abs_k,j] is in block_i_j_shared[k,threadIdx.y]
        int using_k_path = sum_if_not_infinite(
            block_t_t_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, k, blockDim.x)], 
            block_i_j_shared[ARR_MATRIX_INDEX(k, threadIdx.y, blockDim.x)], 
            INF
        ); 

        if (using_k_path < block_i_j_shared[ARR_MATRIX_INDEX(threadIdx.x, threadIdx.y, blockDim.x)]) {
            block_i_j_shared[ARR_MATRIX_INDEX(threadIdx.x, threadIdx.y, blockDim.x)] = using_k_path;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();
    }

    // copy result in global memory
    matrix[ARR_MATRIX_INDEX(i, j, n)] = block_i_j_shared[ARR_MATRIX_INDEX(threadIdx.x, threadIdx.y, blockDim.x)];
}


__global__ void execute_round_device_v_3_1_phase_2_col_portion(int *matrix, int n, int t, int start_row, int end_row) {
    
    if (blockIdx.x >= end_row-start_row)    return;
    
    extern __shared__ int shared_mem[];

    int* block_i_j_shared = &shared_mem[0];
    int* block_t_t_shared = &shared_mem[blockDim.x*blockDim.x];

    // it's a column ...

    // abs row index 
    int i = BLOCK_START(blockIdx.x, blockDim.x) + threadIdx.x + start_row * blockDim.x;
    // abs col index 
    int j = BLOCK_START(t, blockDim.x) + threadIdx.y;

    // the block where I am working (transposed to avoid bank conflict)
    block_i_j_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, threadIdx.y, blockDim.x)] = matrix[ARR_MATRIX_INDEX(i, j, n)];

    // the self-dependent block already calculated in this round 
    block_t_t_shared[ARR_MATRIX_INDEX(threadIdx.x, threadIdx.y, blockDim.x)] = matrix[
        ARR_MATRIX_INDEX(
            (BLOCK_START(t, blockDim.x) + threadIdx.x), 
            (BLOCK_START(t, blockDim.x) + threadIdx.y), 
            n
        )
    ];
    
    __syncthreads();

    // now k is iterating the relative indexind of (t,t) block 
    // in shared memory (instead of the abs position in matrix)
    for (int k = 0; k < blockDim.x; k++) {
        
        // Because we are doing columns:
        // -    matrix[i,k] is in block_i_j_shared[threadIdx.x,k]
        // -    matrix[k,j] is in block_t_t_shared[k,threadIdx.y]
        int using_k_path = sum_if_not_infinite(
            block_i_j_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, k, blockDim.x)], 
            block_t_t_shared[ARR_MATRIX_INDEX(k, threadIdx.y, blockDim.x)], 
            INF
        ); 

        if (using_k_path < block_i_j_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, threadIdx.y, blockDim.x)]) {
            block_i_j_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, threadIdx.y, blockDim.x)] = using_k_path;
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();
    }

    // copy result in global memory
    matrix[ARR_MATRIX_INDEX(i, j, n)] = block_i_j_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, threadIdx.y, blockDim.x)];
}



__global__ void execute_round_device_v_3_1_phase_3_portion(int *matrix, int n, int t, int start_row, int start_col, int end_row, int end_col) {

    if (blockIdx.x >= end_row-start_row || blockIdx.y >= end_col-start_col)    return;
    
    extern __shared__ int shared_mem[];

    int* block_i_t_shared = &shared_mem[0];
    int* block_t_j_shared = &shared_mem[blockDim.x*blockDim.x];

    // abs row index
    int i = threadIdx.x + blockIdx.x * blockDim.x + start_row * blockDim.x;
    // abs col index
    int j = threadIdx.y + blockIdx.y * blockDim.y + start_col * blockDim.y;

    // printf("%d,%d\n",i,j);
    
    // since the cell i,j is read and written only by this thread
    // there is no need to copy its value to shared memory we can just us a local variable
    int cell_i_j = matrix[ARR_MATRIX_INDEX(i, j, n)];
        
    // In phase 3 I copy in two portions of my shared memory
    // the block corresponding to (t, this column) and (this row, t). 

    // (this row, t) is transposed to prevent bank conflict

    block_i_t_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, threadIdx.y, blockDim.x)] = matrix[
        ARR_MATRIX_INDEX(i, (BLOCK_START(t, blockDim.x) + threadIdx.y), n)
    ];
    block_t_j_shared[ARR_MATRIX_INDEX(threadIdx.x, threadIdx.y, blockDim.x)] = matrix[
        ARR_MATRIX_INDEX((BLOCK_START(t, blockDim.x) + threadIdx.x), j, n)
    ];
    
    __syncthreads();

    // now k is iterating the relative indexind of (t,t) block 
    // in shared memory (instead of the abs position in matrix)
    for (int k = 0; k < blockDim.x; k++) {

        int using_k_path = sum_if_not_infinite(
            block_i_t_shared[ARR_MATRIX_INDEX_TRASP(threadIdx.x, k, blockDim.x)],
            block_t_j_shared[ARR_MATRIX_INDEX(k, threadIdx.y, blockDim.x)],
            INF
        ); 

        if (using_k_path < cell_i_j) {
            cell_i_j = using_k_path;
        }

        __syncthreads();
    }

    // copy result in global memory
    matrix[ARR_MATRIX_INDEX(i, j, n)] = cell_i_j;
}
