
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>

#include "../include/adj_matrix_utils.hpp"
#include "../include/performance_test.cuh"

void do_nvprof_performance_test(void (*floyd_warshall_arr_algorithm)(int* matrix, int n, int B), int input_size, int blocking_factor, int number_of_tests, int seed) {

    int* arr_matrix = allocate_arr_matrix(input_size, input_size);

    for (int i=0; i<number_of_tests; i++) {

        populate_arr_adj_matrix(arr_matrix, input_size, seed*(i+1), false);

        hipProfilerStart();
        floyd_warshall_arr_algorithm(arr_matrix, input_size, blocking_factor);
        hipProfilerStop();

        printf("Performed test number %d\n", i);
    }
}