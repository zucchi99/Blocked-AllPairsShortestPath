#include "../include/cuda_errors_utils.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"

void handle_error(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit(EXIT_FAILURE);
    }
}

void check_CUDA_error(const char *msg) {
    hipError_t err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf(stderr, "ERRORE CUDA: >%s<: >%s<. Eseguo: EXIT\n", msg, hipGetErrorString(err) );
        exit(-1);
    }
}