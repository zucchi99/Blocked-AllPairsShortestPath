#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "../include/cuda_errors_utils.cuh"

void handle_error(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit(EXIT_FAILURE);
    }
}

void check_CUDA_error(const char *msg) {
    hipError_t err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf(stderr, "ERRORE CUDA: >%s<: >%s<. Eseguo: EXIT\n", msg, hipGetErrorString(err) );
        exit(-1);
    }
}