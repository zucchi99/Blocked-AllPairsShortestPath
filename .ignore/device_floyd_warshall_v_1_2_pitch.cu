#include "hip/hip_runtime.h"
#include "../include/include_needed_libraries.cuh"

//main device code
void floyd_warshall_blocked_device_v_1_3_pitch(int *matrix, int n, int B);

//rounds code
__global__ void execute_round_device_v_1_2_phase_1(int *matrix, int n, int t, int B, size_t pitch);
__global__ void execute_round_device_v_1_3_phase_2(int *matrix, int n, int t, int B, size_t pitch);
__global__ void execute_round_device_v_1_3_phase_3(int *matrix, int n, int t, int B, size_t pitch);

int main(int argc, char *argv[]) {

    //do_nvprof_performance_test(&floyd_warshall_blocked_device_v_1_3_pitch, 50, 10, 1, time(NULL));
    int n = 10;
    int b = 2;
    int* matrix = allocate_arr_matrix(n, n);
    populate_arr_adj_matrix(matrix, n, time(NULL), true);
    floyd_warshall_blocked_device_v_1_3_pitch(matrix, n, b);
    return 0;

    //return handle_arguments_and_execute(argc, argv, (void(*) (int*, int, int)) &floyd_warshall_blocked_device_v_1_3_pitch);

}

void floyd_warshall_blocked_device_v_1_3_pitch(int *matrix, int n, int B) {

    assert(n%B == 0);                       // B must divide n
    assert(B*B<=MAX_BLOCK_SIZE);            // B*B cannot exceed mmax block size

    int *dev_rand_matrix;
    size_t pitch;                          //size in bytes of memory allocated to guarantee alignment
    size_t width = n * sizeof(int);
    size_t height = n;

    //hipMallocPitch(&devPtr, &devPitch, N_cols * sizeof(type), N_rows);

    HANDLE_ERROR(hipMallocPitch( (void**) &dev_rand_matrix, &pitch, width, height));
    //HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n * n * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy2D(dev_rand_matrix, pitch, matrix, width, width, height, hipMemcpyHostToDevice));

    int num_rounds = n/B;
     
    for(int t = 0; t < num_rounds; t++) { 

        printf("round %d of %d\n", t, num_rounds);
        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        dim3 num_blocks_phase_1(1, 1);
        dim3 threads_per_block_phase_1(B, B);

        printf("1 start\n");

        execute_round_device_v_1_2_phase_1<<<num_blocks_phase_1, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B, pitch);
        HANDLE_ERROR(hipDeviceSynchronize());
        
        printf("1 end\n");

        // phase 2: all blocks that share a row or a column with the self dependent, so
        //  -   all blocks just above or under t
        //  -   all block at left and at right of t


        dim3 num_blocks_phase_2(2, num_rounds-1);  

        printf("2 start\n");

        execute_round_device_v_1_3_phase_2<<<num_blocks_phase_2, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B, pitch);
        HANDLE_ERROR(hipDeviceSynchronize());
        
        printf("2 end\n");

        // phase 3: all the remaining blocks, so all the blocks that don't share a row or a col with t

        dim3 num_blocks_phase_3(num_rounds-1, num_rounds-1); 

        printf("3 start\n");

        execute_round_device_v_1_3_phase_3<<<num_blocks_phase_3, threads_per_block_phase_1>>>(dev_rand_matrix, n, t, B, pitch);
        HANDLE_ERROR(hipDeviceSynchronize()); 
        
        printf("3 end\n");
    }

    // HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy2D(matrix, width, dev_rand_matrix, pitch, width, height, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(dev_rand_matrix));
}


__global__ void execute_round_device_v_1_2_phase_1(int *matrix, int n, int t, int B, size_t pitch) {

    // Launched block and correspondent position in the matrix

    //  t

    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 
    //  .   .   .   t   .   .
    //  .   .   .   .   .   . 
    //  .   .   .   .   .   . 

    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

    int i = tid_x + t * B;  // row
    int j = tid_y + t * B;  // col

    int *cell_i_j = pitched_pointer(matrix, i, j, pitch);

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int* cell_k_j = pitched_pointer(matrix, k, j, pitch); //(int *)((char*) matrix + k * pitch) + j;
        int* cell_i_k = pitched_pointer(matrix, i, k, pitch); //(int *)((char*) matrix + i * pitch) + k;
    
        int using_k_path = sum_if_not_infinite(*cell_i_k, *cell_k_j, INF); 

        if (using_k_path < *cell_i_j) {
            *cell_i_j = using_k_path;
        }
        
        __syncthreads();
    }
}

__global__ void execute_round_device_v_1_3_phase_2(int *matrix, int n, int t, int B, size_t pitch) {

    // Launched blocks and correspondent position in the matrix
    //  -   blockIdx.x says if I am iterating row or cols, 
    //  -   blockIdx.y says something about which row or col)
    //  -   threadIdx.x and threadIdx.y are relative position of cell in block

    //  L1  L2  L3  R1  R2
    //  U1  U2  U3  D1  D2

    //  .   .   .   U1  .   .
    //  .   .   .   U2  .   .
    //  .   .   .   U3  .   .
    //  L1  L2  L3  -   R1  R2
    //  .   .   .   D1  .   .
    //  .   .   .   D2  .   .

    int i, j;

    if (blockIdx.x == 0) {  

        // it's a row ...
        i = BLOCK_START(t, B) + threadIdx.x;

        if (blockIdx.y < t) {

            // ... and it's the left one
            j = BLOCK_START(blockIdx.y, B) + threadIdx.y;

        } else {
            
            // ... and it's the right one
            j = BLOCK_START(blockIdx.y, B) + B + threadIdx.y;
        }
    } else {

        // it's a column ...
        j = BLOCK_START(t, B) + threadIdx.y;

        if (blockIdx.y < t) {

            // ... and it's the up one
            i = BLOCK_START(blockIdx.y, B) + threadIdx.x;

        } else {

            // ... and it's the down one
            i = BLOCK_START(blockIdx.y, B) + B + threadIdx.x;
        }
    }

    int *cell_i_j = pitched_pointer(matrix, i, j, pitch); 

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        if (
            /* row index is contained in s.d. block and column index is outside */
            ( BLOCK_START(t,B)<=i<BLOCK_END(t,B) && (j<BLOCK_START(t,B) || j>=BLOCK_END(t,B)) )   ||  

            /* column index is contained in s.d. block and row index is outside */
            ( BLOCK_START(t,B)<=j<BLOCK_END(t,B) && (i<BLOCK_START(t,B) || i>=BLOCK_END(t,B)) ) 
            ) {

            int* cell_k_j = pitched_pointer(matrix, k, j, pitch); //(int *)((char*) matrix + k * pitch) + j;
            int* cell_i_k = pitched_pointer(matrix, i, k, pitch); //(int *)((char*) matrix + i * pitch) + k;
    
            int using_k_path = sum_if_not_infinite(*cell_i_k, *cell_k_j, INF); 
    
            if (using_k_path < *cell_i_j) {
                *cell_i_j = using_k_path;
            }
        }

        //printf("i:%d, j:%d, k:%d\n", i, j, k);

        __syncthreads();

    }
}


__global__ void execute_round_device_v_1_3_phase_3(int *matrix, int n, int t, int B, size_t pitch) {

    // Launched blocks and correspondent position in the matrix

    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  UL  UL  UL  UR  UR
    //  DL  DL  DL  DR  DR
    //  DL  DL  DL  DR  DR

    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR
    //  UL  UL  UL  -   UR  UR  
    //  -   -   -   -   -   - 
    //  DL  DL  DL  -   DR  DR
    //  DL  DL  DL  -   DR  DR

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
           
    // if a thread is under t, add B as row offset to get right position in matrix
    if (blockIdx.x >= t)    i += B; 

    // if a thread is ar right of t, add B as col offset to get right position in matrix
    if (blockIdx.y >= t)    j += B;

    int *cell_i_j = pitched_pointer(matrix, i, j, pitch); 

    //foreach k: t*B <= t < t+B
    for (int k = BLOCK_START(t,B); k < BLOCK_END(t,B); k++) {

        int* cell_k_j = pitched_pointer(matrix, k, j, pitch); //(int *)((char*) matrix + k * pitch) + j;
        int* cell_i_k = pitched_pointer(matrix, i, k, pitch); //(int *)((char*) matrix + i * pitch) + k;

        int using_k_path = sum_if_not_infinite(*cell_i_k, *cell_k_j, INF); 

        if (using_k_path < *cell_i_j) {
            *cell_i_j = using_k_path;
        }

        __syncthreads();
    }
}

