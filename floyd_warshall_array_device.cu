#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>

#include <ctime>


#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "include/adj_matrix_utils.hpp"
#include "include/host_floyd_warshall.hpp"
#include "include/cuda_errors_utils.cuh"
#include "include/performance_test.cuh"
#include "include/statistical_test.hpp"


void floyd_warshall_blocked_device_v1_0(int *matrix, int n, int B);
__global__ void execute_round_device(int *matrix, int n, int t, int row, int col, int B);
// void temp_statistical_test(int n_tests, size_t input_size, int BLOCKING_FACTOR, bool stop_if_fail);


int main() {

    // for (size_t n = 10; n < 200; n += 2) {

    //     int MAX_B = min(32, n);
    
    //     for (int BLOCKING_FACTOR = 1; BLOCKING_FACTOR < MAX_B; BLOCKING_FACTOR += 2) {
    //         if((n % BLOCKING_FACTOR) == 0) {
                
    //             printf("n: %ld, B: %d\n", n, BLOCKING_FACTOR);
    //             do_arr_floyd_warshall_statistical_test(&floyd_warshall_blocked_device_v1_0, n, BLOCKING_FACTOR, 1000, RANDOM_SEED, true, 4);

    //         }
    //     }
    // }

    int n = 128;
    int b = 32;
    int n_tests = 1000;
    int seed = 2862999;
    // int seed = RANDOM_SEED;

    do_arr_floyd_warshall_statistical_test(&floyd_warshall_blocked_device_v1_0, n, b, n_tests, seed, false, 4, false);

    // 
    // do_nvprof_performance_test(&floyd_warshall_blocked_device_v1_0, n, BLOCKING_FACTOR, 100, clock());
    

    // int *input_instance = (int *) malloc(sizeof(int *) * n * n);
    // int *test_instance_space = (int *) malloc(sizeof(int *) * n * n);
    // populate_arr_graph(input_instance, n, seed);
    // copy_arr_graph(input_instance, test_instance_space, n);
    // bool result = test_arr_floyd_warshall(&floyd_warshall_blocked_device_v1_0, input_instance, test_instance_space, n, b);
    // printf("Corretto: %s\n", bool_to_string(result));

    return 0;
}

__global__ void execute_round_device(int *matrix, int n, int t, int row, int col, int B) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int i = tid/n;  // row
    int j = tid%n;  // col

    //foreach k: t*B <= t < t+B
    for (int k = t * B; k < (t+1) * B; k++) {

        int a, b;

        // check if thread correspond to one of the cells in current block
        if (i>=row * B && i<(row+1) * B && j>=col * B && j<(col+1) * B) {

            // WARNING: do NOT put the macro directly into 
            a = matrix[i*n + j];
            b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 
        }

        __syncthreads();


        if (i>=row * B && i<(row+1) * B && j>=col * B && j<(col+1) * B) {
            matrix[i*n + j] = min(a, b);
        }
        
        __syncthreads();

    }
}



void floyd_warshall_blocked_device_v1_0(int *matrix, int n, int B) {

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));
    
    int num_rounds = n/B;

    int num_blocks = num_rounds*num_rounds;
    int thread_per_block = B*B; 
    

    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, t, B);

        //phase 2 blocks left
        for (int j = t-1; j >= 0; j--) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
        }

        //phase 2 blocks above
        for (int i = t-1; i >= 0; i--) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
        }

        //phase 2 blocks below
        for (int i = t+1; i < num_rounds; i++) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
        }

        //phase 2 blocks right
        for (int j = t+1; j < num_rounds; j++) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
        }
        
        //phase 2,3: remaining blocks
        //phase 3 blocks above and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }
        //phase 3 blocks above and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }
        //phase 3 blocks below and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }      
        //phase 3 blocks below and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }   
    }

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_rand_matrix));
}