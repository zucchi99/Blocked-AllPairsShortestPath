#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>

#include <ctime>


#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "include/adj_matrix_utils.hpp"
#include "include/host_floyd_warshall.hpp"
#include "include/cuda_errors_utils.cuh"
#include "include/performance_test.cuh"


void floyd_warshall_blocked_device_v1_0(int *matrix, int n, int B);
__global__ void execute_round_device(int *matrix, int n, int t, int row, int col, int B);


void temp_statistical_test(int n_tests, size_t input_size, int BLOCKING_FACTOR, bool stop_if_fail) {

    int n_wrong = 0;

    for (size_t i = 0; i < n_tests; i++)
    {
        //random seed
        int rand_seed = clock(); // (i+1)*clock(); //time(NULL);
        // srand(rand_seed);
        printf("%lu/%d)\tseed: %d", i, n_tests, rand_seed);

        //matrix initialization
        int *rand_matrix_1 = (int *) malloc(sizeof(int *) * input_size * input_size);
        int *rand_matrix_2 = (int *) malloc(sizeof(int *) * input_size * input_size);
        populate_arr_graph(rand_matrix_1, input_size, rand_seed);
        copy_arr_graph(rand_matrix_1, rand_matrix_2, input_size);
        // generate_arr_graph(input_size, rand_seed);

        //floyd_warshall execution
        arr_floyd_warshall(rand_matrix_1, input_size);

        //---------------------------------------------------------------

        //matrix initialization with same seed
        //  int *rand_matrix_2 = generate_arr_graph(input_size, rand_seed);
        
        //floyd_warshall_blocked execution (on device)
        floyd_warshall_blocked_device_v1_0(rand_matrix_2, input_size, BLOCKING_FACTOR);
        // arr_floyd_warshall_blocked(rand_matrix_2, input_size, BLOCKING_FACTOR);
        
        //---------------------------------------------------------------

        //compare matrixes output
        bool are_the_same = same_arr_matrix(rand_matrix_1, rand_matrix_2, input_size);

        if (!are_the_same) {

            n_wrong++;

            // //matrix print
            // printf("\ninput adjacency matrix %lux%lu:\n", input_size, input_size);
            // print_arr_matrix(rand_matrix_1, input_size, input_size);

            // //print floyd_warshall output
            // printf("output adjacency matrix classic %lux%lu:\n", input_size, input_size);
            // print_arr_matrix(rand_matrix_1, input_size, input_size);

            // //print floyd_warshall_blocked output
            // printf("output adjacency matrix blocked %lux%lu:\n", input_size, input_size);
            // print_arr_matrix(rand_matrix_2, input_size, input_size);
            // printf("Matrixes are equal? %s\n", bool_to_string(are_the_same));

            printf(" ERROR!\n"); 
            
            if (stop_if_fail) break;
        } else {
            printf("\tOK!\n");
        }

        free(rand_matrix_1);
        free(rand_matrix_2);
    }

    printf("%d errors detected\n\n", n_wrong);
}


int main() {

    //matrix size n*n
    size_t n = 128;

    //if no weights in graph:
    //int INF = (n * (n-1) / 2) + 1;

    int BLOCKING_FACTOR = 16;

    // temp_statistical_test(10000, n, BLOCKING_FACTOR, true);
    // do_nvprof_performance_test(&floyd_warshall_blocked_device_v1_0, n, BLOCKING_FACTOR, 100, clock());
    
    return 0;
}

__global__ void execute_round_device(int *matrix, int n, int t, int row, int col, int B) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int i = tid/n;  // row
    int j = tid%n;  // col

    //foreach k: t*B <= t < t+B
    for (int k = t * B; k < (t+1) * B; k++) {

        int a, b;

        // check if thread correspond to one of the cells in current block
        if (i>=row * B && i<(row+1) * B && j>=col * B && j<(col+1) * B) {

            // WARNING: do NOT put the macro directly into 
            a = matrix[i*n + j];
            b = sum_if_not_infinite(matrix[i*n + k], matrix[k*n + j], INF); 
        }

        __syncthreads();


        if (i>=row * B && i<(row+1) * B && j>=col * B && j<(col+1) * B) {
            matrix[i*n + j] = min(a, b);
        }
        
        __syncthreads();

    }
}



void floyd_warshall_blocked_device_v1_0(int *matrix, int n, int B) {

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));
    
    int num_rounds = n/B;

    int num_blocks = num_rounds*num_rounds;
    int thread_per_block = B*B; 
    

    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, t, B);

        //phase 2 blocks left
        for (int j = t-1; j >= 0; j--) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
        }

        //phase 2 blocks above
        for (int i = t-1; i >= 0; i--) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
        }

        //phase 2 blocks below
        for (int i = t+1; i < num_rounds; i++) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
        }

        //phase 2 blocks right
        for (int j = t+1; j < num_rounds; j++) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
        }
        
        //phase 2,3: remaining blocks
        //phase 3 blocks above and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }
        //phase 3 blocks above and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }
        //phase 3 blocks below and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }      
        //phase 3 blocks below and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }   
    }

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_rand_matrix));
}