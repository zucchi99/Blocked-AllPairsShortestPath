#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>

#include "hip/hip_runtime.h"
#include ""

#include "include/adj_matrix_utils.hpp"
#include "include/host_floyd_warshall.hpp"

//error handling (todo: copy
// #define HANDLE_ERROR(err) (handle_error(err, __FILE__, __LINE__))
// static void handle_error(hipError_t err, const char *file, int line);
// void check_CUDA_error(const char *msg);

#include "include/cuda_errors_utils.cuh"


void floyd_warshall_blocked_device_v1_0(int *matrix, int n, int B);
__global__ void execute_round_device(int *matrix, int n, int t, int row, int col, int B);


int main() {

    //matrix size n*n
    size_t n = 6;

    //if no weights in graph:
    //int INF = (n * (n-1) / 2) + 1;

    int BLOCKING_FACTOR = 2;

    //memory allocation 
    // int *rand_matrix_1 = (int *) malloc(sizeof(int *) * n * n);
    // int *rand_matrix_2 = (int *) malloc(sizeof(int *) * n * n);

    int n_wrong = 0;

    for (size_t i = 0; i < 25; i++)
    {
        //random seed
        int rand_seed = i*time(NULL);
        // srand(rand_seed);
        printf("seed: %d", rand_seed);

        //matrix initialization
        int *rand_matrix_1 = generate_arr_graph(n, rand_seed);

        //floyd_warshall execution
        arr_floyd_warshall(rand_matrix_1, n);

        //---------------------------------------------------------------

        //matrix initialization with same seed
        int *rand_matrix_2 = generate_arr_graph(n, rand_seed);
        
        //floyd_warshall_blocked execution (on device)
        floyd_warshall_blocked_device_v1_0(rand_matrix_2, n, BLOCKING_FACTOR);
        // arr_floyd_warshall_blocked(rand_matrix_2, n, BLOCKING_FACTOR);
        
        //---------------------------------------------------------------

        //compare matrixes output
        bool are_the_same = same_arr_matrix(rand_matrix_1, rand_matrix_2, n);

        if (!are_the_same) {

            n_wrong++;

            //matrix print
            printf("\ninput adjacency matrix %lux%lu:\n", n, n);
            print_arr_matrix(rand_matrix_1, n, n);

            //print floyd_warshall output
            printf("output adjacency matrix classic %lux%lu:\n", n, n);
            print_arr_matrix(rand_matrix_1, n, n);

            //print floyd_warshall_blocked output
            printf("output adjacency matrix blocked %lux%lu:\n", n, n);
            print_arr_matrix(rand_matrix_2, n, n);
            printf("Matrixes are equal? %s\n", bool_to_string(are_the_same));
        } else {
            printf("\tOK!\n");
        }

        free(rand_matrix_1);
        free(rand_matrix_2);
    }

    printf("%d errors detected\n\n", n_wrong);
    
    return 0;
}

// static void handle_error(hipError_t err, const char *file, int line) {
//     if (err != hipSuccess) {
//         printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
//         exit(EXIT_FAILURE);
//     }
// }

// void check_CUDA_error(const char *msg) {
//     hipError_t err = hipGetLastError();
//     if(hipSuccess != err) {
//         fprintf(stderr, "ERRORE CUDA: >%s<: >%s<. Eseguo: EXIT\n", msg, hipGetErrorString(err) );
//         exit(-1);
//     }
// }

__global__ void execute_round_device(int *matrix, int n, int t, int row, int col, int B) {
    
    //foreach k: t*B <= t < t+B
    int block_start = t * B;
    int block_end = (t+1) * B;

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int i = tid/n;  // row
    int j = tid%n;  // col

    // printf("(%d, %d)\n", i, j);

    int row_start = row * B;
    int row_end = (row+1) * B;
    int col_start = col * B;
    int col_end = (col+1) * B;


    for (int k = block_start; k < block_end; k++) {

        int a, b, x1, x2;

        if (i>=row_start && i<row_end && j>=col_start && j<col_end) {

            x1 = matrix[i*n + k];
            x2 =  matrix[k*n + j];
            a = matrix[i*n + j];
            b = sum_if_not_infinite(x1,x2, INF);
        }

        __syncthreads();

        if (i>=row_start && i<row_end && j>=col_start && j<col_end) {

            matrix[i*n + j] = min(a, b); 
        }

        __syncthreads();
    }
}



void floyd_warshall_blocked_device_v1_0(int *matrix, int n, int B) {

    int *dev_rand_matrix;
    HANDLE_ERROR(hipMalloc( (void**) &dev_rand_matrix, n * n* sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_rand_matrix, matrix, n*n*sizeof(int), hipMemcpyHostToDevice));
    
    int num_rounds = n/B;

    int num_blocks = num_rounds*num_rounds;
    int thread_per_block = B*B; 
    

    for(int t = 0; t < num_rounds; t++) { 

        //arr_execute_round(int *matrix, int n, int t, int row, int col, int B)

        //phase 1: self-dependent block
        execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, t, B);

        //phase 2 blocks left
        for (int j = t-1; j >= 0; j--) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
        }

        //phase 2 blocks above
        for (int i = t-1; i >= 0; i--) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
        }

        //phase 2 blocks below
        for (int i = t+1; i < num_rounds; i++) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, t, B);
        }

        //phase 2 blocks right
        for (int j = t+1; j < num_rounds; j++) {
            execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, t, j, B);
        }
        
        //phase 2,3: remaining blocks
        //phase 3 blocks above and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }
        //phase 3 blocks above and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t-1; i >= 0; i--) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }
        //phase 3 blocks below and left
        for (int j = t-1; j >= 0; j--) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }      
        //phase 3 blocks below and right
        for (int j = t+1; j < num_rounds; j++) {
            for (int i = t+1; i < num_rounds; i++) {
                execute_round_device<<<num_blocks, thread_per_block>>>(dev_rand_matrix, n, t, i, j, B);
            }
        }   
    }

    HANDLE_ERROR(hipMemcpy(matrix, dev_rand_matrix, n*n*sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_rand_matrix));
}